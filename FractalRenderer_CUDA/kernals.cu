#include "hip/hip_runtime.h"
#include "kernals.cuh"

__global__ void mandlebrot(size_t w, size_t h, size_t it, double x_min, double y_min, double x_scale, double y_scale, double* data) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	size_t y = idx / w;
	size_t x = idx % w;

    if (y < h) {
        double fx = x_min + ((double)x * x_scale);
        double fy = y_min + ((double)y * y_scale);

        hipDoubleComplex c = make_hipDoubleComplex(fx, fy);
        hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);

        for (size_t i = 0; i < it; i++) {
            z = hipCfma(z, z, c);

            if (hipCabs(z) > 2.0) {
                data[idx] = 1.0 - (1.0 / (((double)i / 50.00) + 1.0));
                return;
            }
        }

        data[idx] = 1.0;

       /* size_t i = 0;
        for (; i < it; i++) {
            z = hipCfma(z, z, c);

            if (hipCabs(z) > 2.0) {
                break;
            }
        }

        if (i < it) {
            double smooth_color = i + 1.0 - log(log(hipCabs(z))) / log(2.0);
            data[idx] = smooth_color / it;
        }
        else {
            data[idx] = 0.0;
        }*/

    }
}